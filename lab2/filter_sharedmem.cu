#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4
// 2022-12-07: A correction for a deprecated function.

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10

const int blockSize = 32; // #threads per block

/* This is the original code -----------
__global__ void filter(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{ 
  // map from blockIdx to pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int dy, dx;
  	unsigned int sumx, sumy, sumz;

  	int divby = (2*kernelsizex+1)*(2*kernelsizey+1); // Works for box filters only!
	
	if (x < imagesizex && y < imagesizey) // If inside image
	{
// Filter kernel (simple box filter)
	sumx=0;sumy=0;sumz=0;
	for(dy=-kernelsizey;dy<=kernelsizey;dy++)
		for(dx=-kernelsizex;dx<=kernelsizex;dx++)	
		{
			// Use max and min to avoid branching!
			int yy = min(max(y+dy, 0), imagesizey-1);
			int xx = min(max(x+dx, 0), imagesizex-1);
			
			sumx += image[((yy)*imagesizex+(xx))*3+0];
			sumy += image[((yy)*imagesizex+(xx))*3+1];
			sumz += image[((yy)*imagesizex+(xx))*3+2];
		}
	out[(y*imagesizex+x)*3+0] = sumx/divby;
	out[(y*imagesizex+x)*3+1] = sumy/divby;
	out[(y*imagesizex+x)*3+2] = sumz/divby;
	}
} 
*/

// function for the filter with shared memory
__global__ void filter_sharedmem(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{  
    // map from blockIdx to pixel position
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// normalization factor
	int divby = (2*kernelsizex+1)*(2*kernelsizey+1); // Works for box filters only!

    // int dy, dx;
    unsigned int sumx, sumy, sumz;

	/* ---Shared memeoty part-- */
    // Max #Threads per block (3 chars per pixel)
    __shared__ unsigned char imgArray[blockSize*blockSize*3];
  
    imgArray[(threadIdx.y*blockDim.x+threadIdx.x)*3+0] = image[(y*imagesizex+x)*3+0];
    imgArray[(threadIdx.y*blockDim.x+threadIdx.x)*3+1] = image[(y*imagesizex+x)*3+1];
	imgArray[(threadIdx.y*blockDim.x+threadIdx.x)*3+2] = image[(y*imagesizex+x)*3+2];

    // synchronize between threads 
    __syncthreads();

	if (x < imagesizex && y < imagesizey) // If inside image
	{
        // Filter kernel (simple box filter)
        sumx=0;sumy=0;sumz=0;
        //Loop across image, from -filtersize to +filtersize 
        //Default is 2,2 ( so it is a 5x5 box filter.) -2   -   2 (-2,-1,0,1,2)
        for(int dy=-kernelsizey;dy<=kernelsizey;dy++) {  // y direction
            // both directions, total size is 5x5 = 25 (DivBy) variable
            for(int dx=-kernelsizex;dx<=kernelsizex;dx++) { // x direction
                
				// creating a new indexing variable becuase we not have a shared memory bank
				int index = (threadIdx.y*blockDim.x+threadIdx.x) + dy*blockDim.x + dx; 

				/* --- the seris of checks --- 
				1. Ensuring that the index is withing the block bounds 
				2. The shared memory for one element near and at the corners are ignored
				*/
                if(!((index < 0) || (index > (blockSize*blockSize-1)) || (dx > (blockDim.x-kernelsizex)) || (dy > (blockDim.y-kernelsizey)))) {
                    
                    sumx += imgArray[index*3+0];
                    sumy += imgArray[index*3+1];
                    sumz += imgArray[index*3+2];
                    
                } 
				else { // Outside of block row
                    int yy = min(max(y+dy, 0), imagesizey-1);
                    int xx = min(max(x+dx, 0), imagesizex-1);

                    sumx += image[((yy)*imagesizex+(xx))*3+0];
                    sumy += image[((yy)*imagesizex+(xx))*3+1];
                    sumz += image[((yy)*imagesizex+(xx))*3+2];
                }
            }
        }

        // printf("inverse is %f \n", invDivBy);
        // printf("Inverse x is %f", sumx*invDivBy);
        // printf(" x is %f", sumx/divby);

        out[(y*imagesizex+x)*3+0] = sumx/divby;
        out[(y*imagesizex+x)*3+1] = sumy/divby;
		out[(y*imagesizex+x)*3+2] = sumz/divby;
	}
}


// Global variables for image data

unsigned char *image, *pixels, *dev_bitmap, *dev_input;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// main computation function
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey)
{
	if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY)
	{
		printf("Kernel size out of bounds!\n");
		return;
	}

	pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);

	dim3 dimBlock(blockSize,blockSize);
	dim3 dimgrid(imagesizex/blockSize,imagesizey/blockSize);

	// creating cuda events for timing
	hipEvent_t beforeEvent;
	hipEvent_t afterEvent;
	float theTime;
	hipEventCreate(&beforeEvent);
	hipEventCreate(&afterEvent);
	hipEventRecord(beforeEvent, 0);

	filter_sharedmem<<<dimgrid,dimBlock>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey); // Awful load balance
	hipDeviceSynchronize();

	// cuda timing events
	hipEventRecord(afterEvent, 0);
	hipEventSynchronize(afterEvent);
	hipEventElapsedTime(&theTime, beforeEvent, afterEvent);

	printf("Time to draw: %f ms\n", theTime);

//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
	
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
	hipFree( dev_bitmap );
	hipFree( dev_input );
}

// Display images
void Draw()
{
// Dump the whole picture onto the screen.	
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );

	if (imagesizey >= imagesizex)
	{ // Not wide - probably square. Original left, result right.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
		glRasterPos2i(0, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE,  pixels);
	}
	else
	{ // Wide image! Original on top, result below.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels );
		glRasterPos2i(-1, 0);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
	}
	glFlush();
}

// Main program, inits
int main( int argc, char** argv) 
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );

	if (argc > 1)
		image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
	else
		image = readppm((char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey);

	if (imagesizey >= imagesizex)
		glutInitWindowSize( imagesizex*2, imagesizey );
	else
		glutInitWindowSize( imagesizex, imagesizey*2 );
	glutCreateWindow("Lab 5");
	glutDisplayFunc(Draw);

	ResetMilli();

	computeImages(2, 2);

// You can save the result to a file like this:
//	writeppm("out.ppm", imagesizey, imagesizex, pixels);

	glutMainLoop();
	return 0;
}
