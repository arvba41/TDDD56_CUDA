#include "hip/hip_runtime.h"
// Mandelbrot explorer, based on my old Julia demo plus parts of Nicolas Melot's Lab 1 code.
// CPU only! Your task: Rewrite for CUDA! Test and evaluate performance.

// Compile with:
// gcc interactiveMandelbrot.cpp -shared-libgcc -lstdc++-static  -o interactiveMandelbrot -lglut -lGL
// or
// g++ interactiveMandelbrot.cpp -o interactiveMandelbrot -lglut -lGL

// Your CUDA version should compile with something like
// nvcc -lglut -lGL interactiveMandelbrotCUDA.cu -o interactiveMandelbrotCUDA

// Preliminary version 2014-11-30
// Cleaned a bit more 2014-12-01
// Corrected the missing glRasterPos2i 2014-12-03

#ifdef __APPLE__
#include <OpenGL/gl.h>
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#include <GL/gl.h>
#endif
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


// Select precision here! float or double!
#define MYFLOAT double
#define DIM 512


// Image data
unsigned char	*pixels = NULL;
unsigned char	*devicePixels;
// User controlled parameters
int maxiter = 20;
MYFLOAT offsetx = -200, offsety = 0, zoom = 0;
MYFLOAT scale = 1.5;
int imageWidth;
int imageHeight;



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
         fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
         if (abort) exit(code);
      }
}


// Init image data
void initBitmap(int width, int height) {
	if (pixels)
		free(pixels);

	if (devicePixels) {
		gpuErrchk( hipFree(devicePixels) );
		printf("Freeing device pixels.\n");
	}

	int size = width * height * 4 * sizeof(unsigned char);
	pixels = (unsigned char*)malloc(size);
	gpuErrchk(hipMalloc((void**)&devicePixels, size));
	gpuErrchk(hipDeviceSynchronize());

	printf("Bitmap initialized.\n");
}


// Complex number class
struct hipComplex {
	MYFLOAT r;
	MYFLOAT i;

	__device__
	hipComplex( MYFLOAT a, MYFLOAT b ) : r(a), i(b)  {}

	__device__
	float magnitude2( void ) {
		return r * r + i * i;
	}

	__device__
	hipComplex operator*(const hipComplex& a) {
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}

	__device__
	hipComplex operator+(const hipComplex& a) {
		return hipComplex(r+a.r, i+a.i);
	}
};


__device__
int mandelbrot(int x, int y, int maxiter, MYFLOAT offsetx, MYFLOAT offsety, MYFLOAT scale, int imageWidth, int imageHeight) {
	MYFLOAT jx = scale * (MYFLOAT)(imageWidth/2 - x + offsetx/scale)/(imageWidth/2);
	MYFLOAT jy = scale * (MYFLOAT)(imageHeight/2 - y + offsety/scale)/(imageHeight/2);

	hipComplex c(jx, jy);
	hipComplex a(jx, jy);

	int i = 0;
	for (i=0; i<maxiter; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return i;
	}

	return i;
}


__global__
void computeFractal(unsigned char *ptr, int maxiter, MYFLOAT offsetx,
                    MYFLOAT offsety, MYFLOAT scale,
                    int imageWidth, int imageHeight) {
	int indexX = blockIdx.x * blockDim.x + threadIdx.x;
	int indexY = blockIdx.y * blockDim.y + threadIdx.y;
	int index = indexY * imageWidth + indexX;

	// Calculate the value at that position;
	int fractalValue = mandelbrot(indexX, indexY, maxiter, offsetx, offsety, scale, imageWidth, imageHeight);

	// Colorize it
	int red = 255 * fractalValue / maxiter;
	if (red > 255) red = 255 - red;
	int green = 255 * fractalValue * 4 / maxiter;
	if (green > 255) green = 255 - green;
	int blue = 255 * fractalValue * 20 / maxiter;
	if (blue > 255) blue = 255 - blue;

	ptr[index * 4 + 0] = red;
	ptr[index * 4 + 1] = green;
	ptr[index * 4 + 2] = blue;
	ptr[index * 4 + 3] = 255;

}


char print_help = 0;

// Yuck, GLUT text is old junk that should be avoided... but it will have to do
static void print_str(void *font, const char *string) {
	int i;

	for (i = 0; string[i]; i++) {
		glutBitmapCharacter(font, string[i]);
	}
}


void PrintHelp() {
	if (print_help)	{
		glPushMatrix();
		glLoadIdentity();
		glOrtho(-0.5, 639.5, -0.5, 479.5, -1.0, 1.0);

		glEnable(GL_BLEND);
		glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
		glColor4f(0.f, 0.f, 0.5f, 0.5f);
		glRecti(40, 40, 600, 440);

		glColor3f(1.f, 1.f, 1.f);
		glRasterPos2i(300, 420);
		print_str(GLUT_BITMAP_HELVETICA_18, "Help");

		glRasterPos2i(60, 390);
		print_str(GLUT_BITMAP_HELVETICA_18, "h - Toggle Help");
		glRasterPos2i(60, 300);
		print_str(GLUT_BITMAP_HELVETICA_18, "Left click + drag - move picture");
		glRasterPos2i(60, 270);
		print_str(GLUT_BITMAP_HELVETICA_18,
							"Right click + drag up/down - unzoom/zoom");
		glRasterPos2i(60, 240);
		print_str(GLUT_BITMAP_HELVETICA_18, "+ - Increase max. iterations by 32");
		glRasterPos2i(60, 210);
		print_str(GLUT_BITMAP_HELVETICA_18, "- - Decrease max. iterations by 32");
		glRasterPos2i(0, 0);

		glDisable(GL_BLEND);

		glPopMatrix();
	}
}


// Compute fractal and display image

void draw() {
	const int blockSize = 8;
	const int imageSize = imageHeight * imageWidth;
	const int size = imageSize * 4 * sizeof(unsigned char);
	dim3 dimBlock(blockSize, blockSize);
	dim3 dimGrid(imageWidth / blockSize, imageHeight / blockSize);

	computeFractal<<<dimGrid, dimBlock>>>(devicePixels, maxiter,
                                        offsetx, offsety, scale,
                                        imageWidth, imageHeight);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(pixels, devicePixels, size, hipMemcpyDeviceToHost));

	// Dump the whole picture onto the screen.
	// (Old-style OpenGL but without lots of geometry that doesn't matter so much.)
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(imageWidth, imageHeight, GL_RGBA, GL_UNSIGNED_BYTE, pixels );

	if (print_help)
		PrintHelp();

	glutSwapBuffers();
}


char explore = 1;

static void Reshape(int width, int height) {
	imageWidth = width;
	imageHeight = height;

	glViewport(0, 0, width, height);
	glLoadIdentity();
	glOrtho(-0.5f, width - 0.5f, -0.5f, height - 0.5f, -1.f, 1.f);
	initBitmap(width, height);

	glutPostRedisplay();
}


int mouse_x, mouse_y, mouse_btn;

// Mouse down
static void mouse_button(int button, int state, int x, int y) {
	if (state == GLUT_DOWN) {
		// Record start position
		mouse_x = x;
		mouse_y = y;
		mouse_btn = button;
	}
}


// Drag mouse
static void mouse_motion(int x, int y) {
	if (mouse_btn == 0) {
		// Ordinary mouse button - move
		offsetx += (x - mouse_x)*scale;
		mouse_x = x;
		offsety += (mouse_y - y)*scale;
		mouse_y = y;

		glutPostRedisplay();
	}
	else {
		// Alt mouse button - scale
		scale *= pow(1.1, y - mouse_y);
		mouse_y = y;
		glutPostRedisplay();
	}
}


void KeyboardProc(unsigned char key, int x, int y) {
	switch (key)	{
	case 27: /* Escape key */
	case 'q':
	case 'Q':
		exit(0);
		break;
	case '+':
		maxiter += maxiter < 1024 - 32 ? 32 : 0;
		break;
	case '-':
		maxiter -= maxiter > 0 + 32 ? 32 : 0;
		break;
	case 'h':
		print_help = !print_help;
		break;
	}
	glutPostRedisplay();
}


// Main program, inits
int main(int argc, char** argv) {
	hipDeviceReset();
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA );
	glutInitWindowSize(DIM, DIM );
	glutCreateWindow("Mandelbrot explorer (GPU)");
	glutDisplayFunc(draw);
	glutMouseFunc(mouse_button);
	glutMotionFunc(mouse_motion);
	glutKeyboardFunc(KeyboardProc);
	glutReshapeFunc(Reshape);

	initBitmap(DIM, DIM);

	glutMainLoop();
}
