#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.
// Update 2022: Changed to hipDeviceSynchronize.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>
#include "milli.c"

const int N = 1024; 
const int blocksize = 1024; 

__global__ 
void add_matrix(float *a_d, float *b_d, float *c_d) 
{
	int Idx = blockIdx.x * blockDim.x + threadIdx.x;
	c_d[Idx] = a_d[Idx] + b_d[Idx];
	
}

int main()
{	
	float *a_h, *b_h, *c_h; // pointer to the host arrays
	float *a_d, *b_d, *c_d; // pointer to the device arrays  

	float thetime; // declating the pointer to store time result
	
	const int size = N*N*sizeof(float);
	
	hipEvent_t myEvent1; //CUDA event 1
	hipEvent_t myEvent2; //CUDA event 2
	
	hipEventCreate(&myEvent1); // CUDA event initialization
	hipEventCreate(&myEvent2); // CUDA event initialization
		
	//allocate arrays on host
	a_h = (float *)malloc(size); 
	b_h = (float *)malloc(size); 
	c_h = (float *)malloc(size); 
	
	hipMalloc((void **) &a_d, size); //allocate arrays on device
	hipMalloc((void **) &b_d, size); //allocate arrays on device
	hipMalloc((void **) &c_d, size); //allocate arrays on device
	
	// initializing the array
	for (int ii = 0; ii < N; ii++) {
		for (int jj = 0; jj < N; jj++) {
			a_h[ii+jj*N] = 10 + (float)ii;
			b_h[ii+jj*N] = (float)jj/N;
		}
	}
	
	// copy the array information from host to device
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice); 
	hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( N, 1 );
	
	hipEventRecord(myEvent1, 0); // inserting event into the cuda stream
	ResetMilli(); // inserting cpu timer (reset)
	
	add_matrix<<<dimGrid, dimBlock>>>(a_d, b_d, c_d);
	
	hipEventRecord(myEvent2, 0); // inserting event into the cuda stream
	int thetime_cpu = GetMicroseconds(); // get cpu time
	
	hipDeviceSynchronize();
	hipEventSynchronize(myEvent2);
	
	hipEventElapsedTime(&thetime, myEvent1, myEvent2);
	
	// copy the array information from device to host
	hipMemcpy( c_h, c_d, size, hipMemcpyDeviceToHost ); 
	
	// free the mallocs
	hipFree( a_d);
	hipFree( b_d);
	hipFree( c_d);
	
	//Data visulaization
	for (int ii = 0; ii < N; ii++) {
		for (int jj = 0; jj < N; jj++) {
			printf("%0.2f ", c_h[ii + jj*N]);
		}
	}
	printf("\n");


	printf("the time taken for the event is %f us GPU event timer. \n", thetime*1000);

	
	printf("the time taken for the event is %d us CPU timer. \n", thetime_cpu);

	printf("done\n");
	return EXIT_SUCCESS;
}
