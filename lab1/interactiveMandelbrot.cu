#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.
// Update 2022: Changed to hipDeviceSynchronize.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#ifdef __APPLE__
#include <OpenGL/gl.h>
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#include <GL/gl.h>
#endif
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

const int N = 32; 
const int blocksize = 64; 

__global__ 
void add_matrix(float *a_d, float *b_d, float *c_d) 
{
	int Idx = blockIdx.y * blockDim.y + threadIdx.y;
	c_d[Idx] = a_d[Idx] + b_d[Idx];
	
}

int main()
{	
	float *a_h, *b_h, *c_h; // pointer to the host arrays
	float *a_d, *b_d, *c_d; // pointer to the device arrays  

	float thetime; // declating the pointer to store time result
	
	const int size = N*N*sizeof(float);
	
	hipEvent_t myEvent1; //CUDA event 1
	hipEvent_t myEvent2; //CUDA event 2
	
	hipEventCreate(&myEvent1); // CUDA event initialization
	hipEventCreate(&myEvent2); // CUDA event initialization
		
	//allocate arrays on host
	a_h = (float *)malloc(size); 
	b_h = (float *)malloc(size); 
	c_h = (float *)malloc(size); 
	
	hipMalloc((void **) &a_d, size); //allocate arrays on device
	hipMalloc((void **) &b_d, size); //allocate arrays on device
	hipMalloc((void **) &c_d, size); //allocate arrays on device
	
	// initializing the array
	for (int ii = 0; ii < N; ii++) {
		for (int jj = 0; jj < N; jj++) {
			a_h[ii+jj*N] = 10 + (float)ii;
			b_h[ii+jj*N] = (float)jj/N;
		}
	}
	
	// copy the array information from host to device
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice); 
	hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice); 
	
	dim3 dimBlock( 1, blocksize );
	dim3 dimGrid( 1, N );
	
	hipEventRecord(myEvent1, 0); // inserting event into the cuda stream
	ResetMilli(); // inserting cpu timer (reset)
	
	add_matrix<<<dimGrid, dimBlock>>>(a_d, b_d, c_d);
	
	hipEventRecord(myEvent2, 0); // inserting event into the cuda stream
	
	hipDeviceSynchronize();
	hipEventSynchronize(myEvent2);
	int thetime_cpu = GetMicroseconds(); // get cpu time
	
	hipEventElapsedTime(&thetime, myEvent1, myEvent2);
	
	// copy the array information from device to host
	hipMemcpy( c_h, c_d, size, hipMemcpyDeviceToHost ); 
	
	// free the mallocs
	hipFree( a_d);
	hipFree( b_d);
	hipFree( c_d);
	
	//Data visulaization
	for (int ii = 0; ii < N; ii++) {
		for (int jj = 0; jj < N; jj++) {
			printf("%0.2f ", c_h[ii + jj*N]);
		}
	}
	printf("\n");


	printf("the time taken for the event is %f us GPU event timer. \n", thetime*1000);

	
	printf("the time taken for the event is %d us CPU timer. \n", thetime_cpu);

	printf("done\n");
	return EXIT_SUCCESS;
}
